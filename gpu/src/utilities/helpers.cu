#include "helpers.h"

#include <iostream>

__device__ unsigned getThIndex() { return blockIdx.x * blockDim.x + threadIdx.x; }

void checkForErrors()
{
    hipDeviceSynchronize();
    printf("Ran into: %s\n",
           hipGetErrorString(hipGetLastError()));
    printf("Sync: %s\n\n", hipGetErrorString(hipDeviceSynchronize()));
}
